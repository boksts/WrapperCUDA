#include "hip/hip_runtime.h"
#include <stdio.h>
#include "hip/hip_runtime.h"
#include ""
#include "MiniWrapForCuda.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#define BLOCK_SIZE 8


//��������������� �������
__global__ void transpose(double* inputMatrix, double* outputMatrix, int width, int height){
	int x = blockDim.x * blockIdx.x + threadIdx.x;
	int y = blockDim.y * blockIdx.y + threadIdx.y;

	if( x < width &&  y < height)
			outputMatrix[x * height + y] = inputMatrix[y * width + x];
}

double *Transp_CUDA(double *a, int N, int M){

	double * a_t = new double[N*M];
	double* a_dev; //�������� ������� 
	double* a_t_dev; //����������������� ������� 
	double* a_t1_dev;

	hipMalloc((void**)&a_dev, N*M * sizeof(double));
	hipMalloc((void**)&a_t_dev, N*M * sizeof(double));

	hipMemcpy(a_dev, a, N*M * sizeof(double), hipMemcpyHostToDevice);

	int gridSizeX = (M / BLOCK_SIZE) + ((M % BLOCK_SIZE) > 0 ? 1 : 0);
	int gridSizeY = (N / BLOCK_SIZE) + ((N % BLOCK_SIZE) > 0 ? 1 : 0);

	dim3 threads(BLOCK_SIZE, BLOCK_SIZE);
	dim3 blocks(gridSizeX, gridSizeY);

	transpose << <blocks, threads >> >(a_dev, a_t_dev, M, N);
	hipDeviceSynchronize();
	
	hipMemcpy(a_t, a_t_dev, N*M * sizeof(double), hipMemcpyDeviceToHost);
	
	hipFree(a_dev);
	hipFree(a_t_dev);

	return a_t;
	delete[] a_t;
}


//��������� ������
__global__ void matMult(double *a, double *b, int M, int N, int Q, double * c)
{
	int   i = blockDim.x * blockIdx.x + threadIdx.x;
	int   j = blockDim.y * blockIdx.y + threadIdx.y;

	if (i < M && j < Q  ){
			c[i*Q + j] = 0;
			for (int k = 0; k < N; k++)
				c[i*Q + j] += a[i*N + k] * b[k*Q + j];
	}
}


double *Mult_CUDA(double *a, double *b, int M, int N, int Q)
{

	double * c = new double[M*Q];

	double * adev = NULL;
	double * bdev = NULL;
	double * cdev = NULL;

	hipMalloc((void**)&adev, sizeof(double) *M*N);
	hipMalloc((void**)&bdev, sizeof(double)*Q*N);
	hipMalloc((void**)&cdev, sizeof(double)*M*Q);

	//����������� ������� �����
	int gridSizeX = (M / BLOCK_SIZE) + ((M % BLOCK_SIZE) > 0 ? 1 : 0);
	int gridSizeY = (Q / BLOCK_SIZE) + ((Q % BLOCK_SIZE) > 0 ? 1 : 0);

	//����������� ����� ������ � �������
	dim3 threads(BLOCK_SIZE, BLOCK_SIZE);
	dim3 blocks(gridSizeX, gridSizeY);

	hipMemcpy(adev, a, sizeof(double) *M*N, hipMemcpyHostToDevice);
	hipMemcpy(bdev, b, sizeof(double) *Q*N, hipMemcpyHostToDevice);

	matMult <<<blocks, threads >>>(adev, bdev, M, N,Q, cdev);
	hipDeviceSynchronize();

	hipMemcpy(c, cdev, sizeof(double) *M*Q, hipMemcpyDeviceToHost);

	hipFree(adev);
	hipFree(bdev);
	hipFree(cdev);


	return c;
	delete c;
}


//��������� ������� �� ������
__global__ void matMultVector(double *a, double *b, int m, int n, double * c)
{
	int tid = blockDim.x * blockIdx.x + threadIdx.x;
	if (tid<m)
	{
		float sum = 0;
		for (int i = 0; i<n; i++)
			sum += a[tid*n + i] * b[i];
		c[tid] = sum;
	}

}


double *MultVector_CUDA(double *a, double *b, int M, int N)
{
	double * c = new double[M];
	double * adev = NULL;
	double * bdev = NULL;
	double * cdev = NULL;

	hipMalloc((void**)&adev, sizeof(double) * N * M);
	hipMalloc((void**)&bdev, sizeof(double) * N);
	hipMalloc((void**)&cdev, sizeof(double) * M);

	//����������� ������� �����
	int gridSizeX = (M / BLOCK_SIZE) + ((M % BLOCK_SIZE) > 0 ? 1 : 0);

	//����������� ����� ������ � �������
	dim3 threads(BLOCK_SIZE);
	dim3 blocks(gridSizeX);

	hipMemcpy(adev, a, sizeof(double) * N * M, hipMemcpyHostToDevice);
	hipMemcpy(bdev, b, sizeof(double) * N, hipMemcpyHostToDevice);

	matMultVector << <blocks, threads >> >(adev, bdev, M, N, cdev);
	hipDeviceSynchronize();

	hipMemcpy(c, cdev, sizeof(double)*M, hipMemcpyDeviceToHost);

	hipFree(adev);
	hipFree(bdev);
	hipFree(cdev);

	return c;
	delete c;
}