#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "MiniWrapForCuda.h"

#define BLOCK_SIZE 64

//�������� ������� � ��������������� ��������, ������� ���������� �� C#
//typedef  double(*FType)(double x);

//��������������� �������
__device__ double func(double x) {
	return (x * x);
}

//��������������� ������� (����� � �����)
__host__ __device__ double funcHost(double x){
	return x*x;
}

//���� ��� �������� 3/8
__global__ void SimpsonMethod_3_8(double* sum_Dev, double* cut_Dev, double a, double b, int n) {

	int i = blockDim.x * blockIdx.x + threadIdx.x;

	double h = (b - a) / n;

	if (i == 0)
		//������ �������� �� ��������
		sum_Dev[i] = ((3.0 / 8.0) * (func(a) + func(b)));
	if (i == 1)
		//������ ��������� �������� �� ��������
		sum_Dev[i] = ((7.0 / 6.0) * (func(a + h) + func(b - h)));
	if (i == 3)
		//������ ��������� �������� �� ��������
		sum_Dev[i] = ((23.0 / 24.0) * (func(a + 2 * h) + func(b - 2 * h)));
	if (i > 3)
		//������ ���������� ��������
		sum_Dev[i] = func(a + (i - 1) * h);
}

//���� ��� ��������
__global__ void SimpsonMethod(double* sum_Dev, double* cut_Dev, double a, double b, int n/*, FType func*/) {

	int i = blockDim.x * blockIdx.x + threadIdx.x;

	double h = (b - a) / n;

	if (i % 2 == 0)
		//������ ������ ���������� ��������
		sum_Dev[i] = 4 * func(a + h * i);
	if (i % 2 == 1)
		//������ �������� ���������� ��������
		sum_Dev[i] = 2 * func(a + h * i);

}

//���� ��� �����
__global__ void GaussMethod(double* sum_Dev, double* xm_Dev, double* cm_Dev, double a, double b, int n, int point) {

	int j = blockDim.x * blockIdx.x + threadIdx.x;

	double h = (b - a) / n;

	for (int i = 0; i < point; i++)
		//������ �������� ��������� �� ����� �����
		sum_Dev[j] += cm_Dev[i] * func(xm_Dev[i] * (h / 2) + a + j * h + h / 2);
}


//������� ��������� ������ � ������ ���� ��� ������ �������� � �������� 3/8
double Compute(float a, float b, int n, void* Function, int method) {

	//�������� ��������� �� ������� �� C#
	//FType F = (FType)(Function);

	double* sum = new double[n];
	double* sum_Dev = NULL;
	double* cut_Dev = NULL;
	double h = (b - a) / n;

	hipMalloc((void**)&sum_Dev, n * sizeof(double));
	hipMalloc((void**)&cut_Dev, n * sizeof(double));

	int gridSizeX = (n / BLOCK_SIZE) + ((n % BLOCK_SIZE) > 0 ? 1 : 0);
	dim3 threads(BLOCK_SIZE, 1);
	dim3 blocks(gridSizeX, 1);

	switch (method) {
	case 1:
		{
			SimpsonMethod << <blocks, threads >>>(sum_Dev, cut_Dev, a, b, n);
			break;
		}
	case 2:
		{
			SimpsonMethod_3_8 << <blocks, threads >> >(sum_Dev, cut_Dev, a, b, n);
			break;
		}
	}

	hipDeviceSynchronize();

	hipMemcpy(sum, sum_Dev, n * sizeof(double), hipMemcpyDeviceToHost);
	hipFree(sum_Dev);
	hipFree(cut_Dev);

	//���������� �������� �����������, ���������� � device
	double result = 0;
	for (int j = 0; j < n; j++)
		result += sum[j];

	if (method == 1)
		result += funcHost(a) + funcHost(b) + 4 * funcHost(a + (n / 2 - 1)*h);

	hipFree(sum_Dev);
	hipFree(cut_Dev);
	delete[] sum;

	switch (method) {
	case 1:
		{
			return (h / 3) * result;
		}
	case 2:
		{
			return h * result;
		}
	}
}

//������������� ������� ��������� ������ � ������ ���� ��� ������ �����
double Compute(float a, float b, int n, void* Function, int method, int point) {
	double h = (b - a) / n;
	double* sum = new double[n];
	double* sum_Dev = NULL;
	double* xm_Dev = NULL;
	double* cm_Dev = NULL;

	double* xm = new double[point];
	double* cm = new double[point];

	//����� �������� ������������� ��� ������� (� ����������� �� ���������� ���������� �����)
	switch (point) {
	case 2:
		xm[0] = -0.5773503;
		xm[1] = -0.5773503;
		cm[0] = 1;
		cm[1] = 1;
		break;
	case 3:
		xm[0] = -0.7745967;
		xm[1] = 0;
		xm[2] = 0.7745967;
		cm[0] = 0.5555556;
		cm[1] = 0.8888889;
		cm[2] = 0.5555556;
		break;
	case 4:
		xm[0] = -0.8611363;
		xm[1] = -0.3399810;
		xm[2] = 0.3399810;
		xm[3] = 0.8611363;
		cm[0] = 0.3478548;
		cm[1] = 0.6521451;
		cm[2] = 0.6521451;
		cm[3] = 0.3478548;
	}

	hipMalloc((void**)&sum_Dev, n * sizeof(double));
	hipMalloc((void**)&xm_Dev, point * sizeof(double));
	hipMalloc((void**)&cm_Dev, point * sizeof(double));

	hipMemcpy(xm_Dev, xm, point * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(cm_Dev, cm, point * sizeof(double), hipMemcpyHostToDevice);

	int gridSizeX = (n / BLOCK_SIZE) + ((n % BLOCK_SIZE) > 0 ? 1 : 0);
	dim3 threads(BLOCK_SIZE, 1);
	dim3 blocks(gridSizeX, 1);

	GaussMethod <<<blocks, threads >>>(sum_Dev, xm_Dev, cm_Dev, a, b, n, point);
	hipDeviceSynchronize();

	hipMemcpy(sum, sum_Dev, n * sizeof(double), hipMemcpyDeviceToHost);

	//���������� �������� �����������, ���������� � device
	double result = 0;
	for (int j = 0; j < n; j++)
		result += sum[j];

	hipFree(sum_Dev);
	hipFree(xm_Dev);
	hipFree(cm_Dev);
	delete[] sum;
	delete[] xm;
	delete[] cm;

	return (h / 2) * result;
}

//����� ��������
//a b  - ��������, n - ����� ���������, * Function - ��������������� �������
double Simpson_CUDA(float a, float b, int n, void* Function) {
	return Compute(a, b, n, Function, 1);
}

//����� �������� 3/8
//a b  - ��������, n - ����� ���������, * Function - ��������������� �������
double Simpson_3_8_CUDA(float a, float b, int n, void* Function) {
	return Compute(a, b, n, Function, 2);
}

//����� �����
//a b  - ��������, n - ����� ���������, * Function - ��������������� �������, point - ����� ����� � ������
double Gauss_CUDA(float a, float b, int n, void* Function, int point) {
	return Compute(a, b, n, Function, 3, point);
}
